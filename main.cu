
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <random>
#include <string>
#include <fstream>
#include <cmath>
#include <assert.h>
#include <cstdio>

#define BLOCK_SIZE 64

#define ADD_TIME(code) do { \
    auto start = std::chrono::system_clock::now(); \
    code \
    auto end   = std::chrono::system_clock::now(); \
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start); \
    time += double(duration.count());\
} while(0)

__global__ void compute_Ap(int n, const float *p, float *Ap){
#define Ap(i, j) Ap[(i) * n + j]
#define p(i, j) p[(i) * n + j]
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        Ap(i, j) = 0.f;
        return;
    }
    Ap(i, j) = 4.0 * p(i, j) - p(i - 1,j) - p(i + 1, j) - p(i, j - 1) - p(i, j + 1);
#undef Ap
#undef p
}

__device__ float warpReduceSum(float val) {
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1){
        val += __shfl_down(val, offset);
    }
    return val;
}

__device__ float blockReduceSum(float val) {
    static __shared__ int shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = warpReduceSum(val);
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if (wid == 0) {
        val = warpReduceSum(val);
    }
    return val;
}

__global__ void deviceReduceKernelStep1(int n, float *p, float *q, float *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float sum = 0;
    for (int i = index; i < n; i += stride) {
        sum += p[i] * q[i];
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x == 0)
        output[blockIdx.x] = sum;
}
__global__ void deviceReduceKernelStep2(int n, float *p, float *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float sum = 0;
    for (int i = index; i < n; i += stride) {
        sum += p[i];
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x == 0)
        output[blockIdx.x] = sum;
}

float reduce(int n, float *p, float *q){
    int blocks = std::min((n * n + BLOCK_SIZE - 1)/ BLOCK_SIZE, 1024);
    auto output = new float[1024];
    deviceReduceKernelStep1<<<blocks, BLOCK_SIZE>>>(n, p, q, output);
    deviceReduceKernelStep2<<<1, 1024>>>(n, output, output);
    return output[0];
}

__global__ void update_x(int n, float *x, const float *p, const float alpha){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride){
        x[i] += alpha * p[i];
    }
}

__global__ void update_r(int n, float *r, const float *p, const float alpha){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride){
        r[i] -= alpha * p[i];
    }
}

__global__ void update_p(int n, const float *r, float *p, const float beta){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride){
        p[i] = r[i] + beta * p[i];
    }
}

__global__ void check_solution(int n, float *Ax, const float *x, const float *b, float *residual){
#define Ax(i, j) Ax[(i) * n + (j)]
#define x(i, j) x[(i) * n + (j)]
#define b(i, j) b[(i) * n + (j)]
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i >= n || j >= n){
        Ax(i, j) = 0.f;
    }
    Ax(i, j) = 4.0 * x(i, j) - x(i - 1, j) - x(i + 1, j) - x(i, j - 1) - x(i, j + 1);
    *residual += (b(i, j) - Ax(i, j)) * (b(i, j) - Ax(i, j));
#undef Ax
#undef x
#undef b
}

float B[2048 * 2048];
float X[2048 * 2048];

void cgSolver(int n, float eps, float *r, float *b, float *x,float *p, float *Ap, float *Ax){
    int size = n * n;
    float alpha = 0.f, beta = 0.f;
    float initial_rTr = reduce(n, r, r);
    printf(">>> Initial residual = %f\n", initial_rTr);
    float old_rTr = initial_rTr;
    update_p<<<n * n / BLOCK_SIZE, BLOCK_SIZE>>>(n, r, p, beta);


    for(int i = 0; i < size; i ++){
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
        compute_Ap<<< dimGrid, dimBlock >>>(n, p, Ap);

        hipDeviceSynchronize();

        float pAp = reduce(n, p, p);
        alpha = old_rTr / pAp;
        update_x<<<(n * n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(n, x, p, alpha);

        hipDeviceSynchronize();

        update_r<<<(n * n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(n, r, p, alpha);

        hipDeviceSynchronize();

        float new_rTr = reduce(n, r, r);

        if (sqrt(new_rTr) < eps){
            printf(">>> Conjugate Gradient method converged at time %d.\n", i);
            break;
        }
        beta = new_rTr / old_rTr;
        update_p<<<(n * n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(n, r, p, beta);
        old_rTr = new_rTr;
    }

    float residual_cg = 0.f;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    check_solution<<<dimGrid, dimBlock >>>(n, Ax, x, b, &residual_cg);

    hipDeviceSynchronize();

    printf(">>> Checking the residual norm(Ax-b)...\n");
    printf(">>> Residual CGPoissonSolver: %f\n",sqrt(residual_cg));
    assert(residual_cg < eps);

    hipMemcpy(X, x, size * sizeof(float), hipMemcpyDeviceToHost);
}

float eps = 1e-8;
int problem_size[5] = {0,256, 512, 1024, 2048};
int repeats = 5;

int main() {
    for(int i = 1; i <= 4; i ++){
        printf("\n>>> Current problem size: %d x %d\n", problem_size[i], problem_size[i]);
        double time = 0.0;
        for(int j = 1; j <= 5; j ++){
            printf(">>> Solving Poisson\'s equation using CG [%d/%d]\n", j, 5);
            int p_size = problem_size[i];
            std::string input_name = "b_" + std::to_string(i) + "_" + std::to_string(p_size) + "_" + std::to_string(j) + ".bin";
            std::ifstream ifs(input_name, std::ios::binary | std::ios::in);
            ifs.read((char *)B, sizeof(float) * p_size * p_size);
            ifs.close();

            int size = p_size * p_size;
            float *r, *b, *x, *p, *Ap, *Ax;
            hipMalloc(&r, size * sizeof(float));
            hipMalloc(&b, size * sizeof(float));
            hipMalloc(&x, size * sizeof(float));
            hipMalloc(&p, size * sizeof(float));
            hipMalloc(&Ap, size * sizeof(float));
            hipMalloc(&Ax, size * sizeof(float));
            hipMemset(&x, 0, size * sizeof(float));
            hipMemset(&p, 0, size * sizeof(float));
            hipMemset(&Ap, 0, size * sizeof(float));
            hipMemset(&Ax, 0, size * sizeof(float));

            hipMemcpy(b, B, size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(r, B, size * sizeof(float), hipMemcpyHostToDevice);

            ADD_TIME(
                    cgSolver(p_size, eps, r, b, x, p, Ap, Ax);
                    );

            hipFree(r);
            hipFree(b);
            hipFree(x);
            hipFree(p);
            hipFree(Ap);
            hipFree(Ax);

            std::string output_name = "ans_" + std::to_string(i) + "_" + std::to_string(p_size) + "_" + std::to_string(j) + ".bin";
            std::ofstream ofs(output_name, std::ios::binary | std::ios::out);
            ofs.write((const char*)X, sizeof(float) * p_size * p_size);
            ofs.close();
        }
        printf("*** Average kernel time: %lf ms\n",time / 5000.0);
    }
}